
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define N 10

float max(float *timer, int n){
	int i = 0;
	float maxTimer=0.0;
	for( ; i < n ; i ++){
		if(timer[i] > maxTimer)maxTimer = timer[i];
	}
	return maxTimer;
}

void flush(float *a, int n){
	int i = 0;
	for( ; i < n ; i++){
		a[i] = 0.0;
	}
}

float sum(float *a, int n){
	float s = 0.0;
	int i = 0;
	for( ; i < N ; i++){
		s += a[i];
	}
	return s;	
}


__global__ void Sort(float *a, int n, float *ts, float *tx, int *signal){
	clock_t start0, finish0;
	clock_t start1, finish1;
	clock_t start2, finish2;

	int blockId = blockIdx.z * (gridDim.x * gridDim.y)
		    + blockIdx.y * gridDim.x
		    + blockIdx.x;
	
	start0 = clock();
	int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
		    + threadIdx.z * (blockDim.x * blockDim.y)
		    + threadIdx.y * blockDim.x
		    + threadIdx.x;
	finish0 = clock();
	ts[threadId] = finish0 - start0;

	float tmp = 0.0;
	float swap = 0.0;
	//even
	if(*signal > 0){
		if(threadId < n){
			if(threadId % 2 == 0){
				start1 = clock();
				tmp = a[threadId] - a[threadId + 1];
				finish1 = clock();
				swap = a[threadId + 1];
				if(tmp <= 0){
					start2 = clock();
					a[threadId + 1] = swap;
					finish2 = clock();
				}
				else{
					a[threadId + 1] = a[threadId];
					start2 = clock();
					a[threadId] = swap;
					finish2 = clock();
				}
				tx[threadId] = (finish2 - start2) + (finish1 - start1);
			}
		}
	}
	//odd
	if(*signal < 0){
		if(threadId + 1 < n){
			if(threadId % 2 == 1){
				start1 = clock();
				tmp = a[threadId] - a[threadId + 1];
				finish1 = clock();
				swap = a[threadId + 1];
				if(tmp <= 0){
					start2 = clock();
					a[threadId + 1] = swap;
					finish2 = clock();
				}
				else{
					a[threadId + 1] = a[threadId];	
					start2 = clock();
					a[threadId] = swap;
					finish2 = clock();
				}
				tx[threadId] = (finish2 - start2) + (finish1 - start1);
			}
		}
	}
}

int main(){
	float overhead = 0.0, TxSum = 0.0, TsSum = 0.0;
	int *signal = (int*)malloc(sizeof(int) * 1);
	float *a = (float*)malloc(sizeof(float) * N);
	float *Tx = (float*)malloc(sizeof(float) * N);
	float *Ts = (float*)malloc(sizeof(float) * N);
	float *ts = (float*)malloc(sizeof(float) * N);
	float *tx = (float*)malloc(sizeof(float) * N);
	signal[0] = 1;
	//intialization array a, Tx, Ts, tx, ts
	int i = 0, x = N-1;
	for( ; i < N ; i++){
		a[i] = (float)x;
		x--;
		Tx[i] = 0.0;
		Ts[i] = 0.0;
		tx[i] = 0.0;
		ts[i] = 0.0;
	}
	
	float *d_a, *d_ts, *d_tx;
	int *d_signal;	
	hipMalloc(&d_a, sizeof(float) * N);
	hipMalloc(&d_tx, sizeof(float) * N);
	hipMalloc(&d_ts, sizeof(float) * N);
	hipMalloc(&d_signal, sizeof(int) * 1);

	hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_tx, tx, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_ts, ts, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_signal, signal, sizeof(int)*1, hipMemcpyHostToDevice);

	dim3 grid(1,1,1);
	dim3 block(N,1,1);
	//GPU 'warm up'
	Sort<<<grid, block>>>(d_a, N, d_ts, d_tx, d_signal);
	
	hipMemcpy(d_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_tx, tx, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_ts, ts, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(d_signal, signal, sizeof(int) * 1, hipMemcpyHostToDevice);
	
	i=0;
	int j = N - 1;	
	for( ; i < N ; i++, j--){

		Sort<<<grid, block>>>(d_a, N, d_ts, d_tx, d_signal);
		
		hipMemcpy(ts, d_ts, sizeof(float) * N, hipMemcpyDeviceToHost);
		Ts[j] = max(ts, N);
		flush(ts, N);
		hipMemcpy(d_ts, ts, sizeof(float) * N, hipMemcpyHostToDevice);	

		hipMemcpy(tx, d_tx, sizeof(float) * N, hipMemcpyDeviceToHost);
		Tx[j] = max(tx, N);
		flush(tx, N);
		hipMemcpy(d_tx, tx, sizeof(float) * N, hipMemcpyHostToDevice);

		*signal = *signal * (-1);
		hipMemcpy(d_signal, signal, sizeof(int) * 1, hipMemcpyHostToDevice);	
	}
	
	hipMemcpy(a, d_a, sizeof(float) * N, hipMemcpyDeviceToHost);
/*
	i = 0;
	for( ; i < N ; i ++){
		printf("a[%d] = %f ; ",i,a[i]);
		printf("\n");
	}
	
	i = 0;
	for( ; i < N ; i ++){
		printf("Tx[%d] = %f ; ",i,Tx[i]);
		printf("\n");
	}
	
	i = 0;
	for( ; i < N ; i ++){
		printf("Ts[%d] = %f ; ",i,Ts[i]);
		printf("\n");
	}
*/
	TxSum = sum(Tx, N);
	TsSum = sum(Ts, N);
	overhead = TxSum + TsSum;
	printf("TxSum = %f, TsSum = %f, overhead = %f \n", TxSum, TsSum, overhead);
	free(a); free(signal); free(Tx); free(Ts); free(ts); free(tx);
	hipFree(d_a); hipFree(d_ts); hipFree(d_tx); hipFree(d_signal);
	
		
	return 0;
}	




















